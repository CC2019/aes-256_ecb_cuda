
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <sys/time.h>


void genRandomString(char *str,int length)
{
  for(int i=0;i<length-1;++i)
  {
    str[i] = 'a' + rand()%26;
  }
  str[length-1] = '\0';
}
void genRandomSubString(char *str,int length,int sub_len)
{
  for(int i=0;i<length-1;++i)
  {
    if(i>0 && ((i+1)%sub_len==0)) str[i] = '\0';
    else str[i] = 'a' + rand()%26;
  }
  str[length-1] = '\0';
}

/*__global__ void my_strstr(char *str,char *sub_string,char ** position,int str_len,int sub_len,int num_sub)
{
  int id = threadIdx.x;
  char *sub = &sub_string[id*sub_len];
  char *string = str;
  char *a,*b;
  b = sub;
  printf("in GPU string is %s sub is %s\n",string,b);
  for(;*string != '\0';++string){
    a = string;
    if(*a == *b){
      printf("thread %d find a possible sub %s\n",id,b);
      while(*(++a) == *(++b)){
        printf("thread %d find a more and more possible sub %s\n",id,b);
        if(*(b+1) == '\0'){ 
          printf("thread %d find a sub %s\n",id,b);
          position[id] = string;
          printf("sting match in %s\n",position[id]);
        }
      }
    }
    b = sub;
  }
}*/

char * my_strstr(char *str,char *sub,int str_len,int sub_len)
{
  if(str_len < sub_len) return NULL;
  if(str_len != 0 && sub_len == 0) return NULL;
  if(str_len == 0 && sub_len == 0) return NULL;
  int m, n;
  for(int i=0;i<str_len;++i){
    m = 0;
    n = i;
    if(str[n]==sub[m]){
      while(str[++n] == sub[++m]){
        if(sub[m+1] == '\0') return str+i;
      }
    }
  }
  return NULL;
}



__global__ void my_strstr(char *str,char *sub_string,char ** position,int str_len,int sub_len,int num_sub)
{
  int id = threadIdx.x; 
  //char *sub = &sub_string[id*sub_len];
  char *result = NULL; 
  char sub[24];
  //load sub in register,great improve
  for(int i=0;i<sub_len;++i){
    sub[i] = sub_string[id*sub_len+i];}
  //best case using Shared memory
  extern __shared__ char s_string[];
  //every thread has to fetch how many values from global memory to shared memory
  int each_num = str_len/blockDim.x;
  for(int i=0;i<each_num;++i){
    s_string[i*blockDim.x+id] = str[i*blockDim.x+id];}
  if( ((each_num*blockDim.x+id) < str_len) && (blockDim.x > each_num) )
    s_string[each_num*blockDim.x+id] = str[each_num*blockDim.x+id];

  char *string = s_string;
  char *a,*b;
  //b point to the sub address in register rather than in global memory
  b = sub;
  //result == NULL to judge if we find a match;rather than use goto or break in loop which harm the calculation
  for(int i = 0;(*string != '\0')&&(result == NULL);i++){
    //printf("i am %d\n",id);
    a = string;
    while(*a++ == *b++){
      if(*(b+1) == '\0'){ 
        result = string;
      }
    }
    b = sub;
    ++string;
  }
  //coalesced global memory store, no effect since we only store once 
  position[id] = result;
}




int main()
{
  int LENGTH = 4096, len = 24, num_sub = 100;
  int num_block,num_thread;
  if(num_sub < 512){
    num_block = 1;
    num_thread = num_sub;
  }
  else{
    num_block = num_sub / 512;
    num_thread = 512;
  }
  char haystack[LENGTH];
  char subs[num_sub*len];
  char *position[num_sub];
  char *h_position[num_sub];
  genRandomString(haystack,LENGTH);
  genRandomSubString(subs,len*num_sub,len);

  char *d_string,*d_subs;
  char **d_position;
  hipMalloc((void**)&d_string,sizeof(char)*LENGTH);
  hipMalloc((void**)&d_subs,sizeof(char)*num_sub*len);
  hipMalloc((void***)&d_position,sizeof(char*)*num_sub);
  hipMemset(d_position,0,sizeof(char*)*num_sub);
  memset(h_position,0,sizeof(char*)*num_sub);
  const size_t smem = sizeof(char)*LENGTH;
  char h_subs[num_sub][len];
  for(int i=0;i<num_sub;++i){
    for(int j=0;j<len;++j){
      h_subs[i][j] = subs[i*len+j];
    }
  }
    
  /*CPU*/

  char *ret;
  struct timeval start,end;
  gettimeofday(&start, NULL );
  for(int i=0;i<num_sub;++i)
  {
    ret = my_strstr(haystack,h_subs[i],LENGTH,len);
    if(ret != NULL){
      printf("find one sub string in %d sub\n",i);
      printf("%s\n",ret);
    }
    position[i] = ret;
  }
  gettimeofday(&end, NULL );
  float timeuse =1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
  printf("CPU time=%f\n",timeuse /1000.0);
  
  /*GPU*/

  gettimeofday(&start, NULL );
  for(int i=0;i<50;++i)
  {
    hipMemcpy(d_string,haystack,sizeof(char)*LENGTH,hipMemcpyHostToDevice);
    hipMemcpy(d_subs,subs,sizeof(char)*num_sub*len,hipMemcpyHostToDevice);
    my_strstr<<<num_block,num_thread,smem>>>(d_string,d_subs,d_position,LENGTH,len,num_sub);
    hipDeviceSynchronize();
    hipMemcpy(h_position,d_position,sizeof(char*)*num_sub,hipMemcpyDeviceToHost);
  }
  gettimeofday(&end, NULL );
  timeuse =1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
  printf("GPU time=%f\n",timeuse /1000.0/50);

  /*check*/

  //in small size GPU works well
  /*for(int i=0;i<num_sub;++i){
    if(h_position[i] == position[i]){
      printf("ok in %d sub\n",i);
      if(position[i] != NULL){
        printf("%s\n",position[i]);
      }
    }
    else{
      printf("error !!!!!!");
      if(position[i] != NULL){
        printf("CPU find match %s\n",position[i]);}
      //because h_position[i] point to the address in GPU , causing segment error
      if(h_position[i] != NULL){
        printf("GPU find match %s\n",h_position[i]);}
    }
  }*/

  return(0);
}
