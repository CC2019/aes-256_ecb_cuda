/*  
*   Byte-oriented AES-256 implementation.
*   All lookup tables replaced with 'on the fly' calculations. 
*
*   Copyright (c) 2007 Ilya O. Levin, http://www.literatecode.com
*
*   Permission to use, copy, modify, and distribute this software for any
*   purpose with or without fee is hereby granted, provided that the above
*   copyright notice and this permission notice appear in all copies.
*
*   THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES
*   WITH REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF
*   MERCHANTABILITY AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR
*   ANY SPECIAL, DIRECT, INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES
*   WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER IN AN
*   ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
*   OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS SOFTWARE.
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#ifndef uint8_t
#define uint8_t  unsigned char
#endif

#ifdef __cplusplus
extern "C" { 
#endif
/*
typedef struct {
  uint8_t key[32]; 
  uint8_t enckey[32]; 
  uint8_t deckey[32];
} aes256_context;
*/

uint8_t ctx_key[32]; 
uint8_t ctx_enckey[32]; 
uint8_t ctx_deckey[32];

__device__ uint8_t sbox_d[256];
__device__ uint8_t sboxinv_d[256];

#ifdef __cplusplus
}
#endif
#define AES_BLOCK_SIZE 16
#define THREADS_PER_BLOCK 512

#define F(x)   (((x)<<1) ^ ((((x)>>7) & 1) * 0x1b))
#define FD(x)  (((x) >> 1) ^ (((x) & 1) ? 0x8d : 0))

const uint8_t sbox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5,
    0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
    0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc,
    0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a,
    0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
    0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b,
    0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
    0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
    0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17,
    0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88,
    0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
    0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9,
    0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6,
    0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
    0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94,
    0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68,
    0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};
const uint8_t sboxinv[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38,
    0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87,
    0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d,
    0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2,
    0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16,
    0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda,
    0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a,
    0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02,
    0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea,
    0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85,
    0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89,
    0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20,
    0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31,
    0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d,
    0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0,
    0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26,
    0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
};

/* -------------------------------------------------------------------------- */
__device__ uint8_t rj_xtime(uint8_t x) 
{
  return (x & 0x80) ? ((x << 1) ^ 0x1b) : (x << 1);
} /* rj_xtime */

/* -------------------------------------------------------------------------- */
__device__ void aes_subBytes(uint8_t *buf)
{
  register uint8_t i, b;
  for (i = 0; i < 16; ++i)
  {
    b = buf[i];
    buf[i] = sbox_d[b];
  }
} /* aes_subBytes */

/* -------------------------------------------------------------------------- */
__device__ void aes_subBytes_inv(uint8_t *buf)
{
  register uint8_t i, b;
  for (i = 0; i < 16; ++i)
  {
    b = buf[i];
    buf[i] = sboxinv_d[b];
  }
} /* aes_subBytes_inv */

/* -------------------------------------------------------------------------- */
__device__ void aes_addRoundKey(uint8_t *buf, uint8_t *key)
{
  register uint8_t i = 16;
  while (i--)
  {
    buf[i] ^= key[i];
  }
} /* aes_addRoundKey */

/* -------------------------------------------------------------------------- */
__device__ void aes_addRoundKey_cpy(uint8_t *buf, uint8_t *key, uint8_t *cpk)
{
  register uint8_t i = 16;
  while (i--)
  {
    buf[i] ^= (cpk[i] = key[i]);
    cpk[16+i] = key[16 + i];
  }
} /* aes_addRoundKey_cpy */

/* -------------------------------------------------------------------------- */
__device__ void aes_shiftRows(uint8_t *buf)
{
  register uint8_t i, j; /* to make it potentially parallelable :) */
  i = buf[1];
  buf[1] = buf[5];
  buf[5] = buf[9];
  buf[9] = buf[13];
  buf[13] = i;
  i = buf[10];
  buf[10] = buf[2];
  buf[2] = i;
  j = buf[3];
  buf[3] = buf[15];
  buf[15] = buf[11];
  buf[11] = buf[7];
  buf[7] = j;
  j = buf[14];
  buf[14] = buf[6];
  buf[6]  = j;
} /* aes_shiftRows */

/* -------------------------------------------------------------------------- */
__device__ void aes_shiftRows_inv(uint8_t *buf)
{
  register uint8_t i, j; /* same as above :) */
  i = buf[1];
  buf[1] = buf[13];
  buf[13] = buf[9];
  buf[9] = buf[5];
  buf[5] = i;
  i = buf[2];
  buf[2] = buf[10];
  buf[10] = i;
  j = buf[3];
  buf[3] = buf[7];
  buf[7] = buf[11];
  buf[11] = buf[15];
  buf[15] = j;
  j = buf[6];
  buf[6] = buf[14];
  buf[14] = j;
} /* aes_shiftRows_inv */

/* -------------------------------------------------------------------------- */
__device__ void aes_mixColumns(uint8_t *buf)
{
  register uint8_t i, a, b, c, d, e;
  for (i = 0; i < 16; i += 4)
  {
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    buf[i] ^= e ^ rj_xtime(a^b);
    buf[i+1] ^= e ^ rj_xtime(b^c);
    buf[i+2] ^= e ^ rj_xtime(c^d);
    buf[i+3] ^= e ^ rj_xtime(d^a);
  }
} /* aes_mixColumns */

/* -------------------------------------------------------------------------- */
__device__ void aes_mixColumns_inv(uint8_t *buf)
{
  register uint8_t i, a, b, c, d, e, x, y, z;
  for (i = 0; i < 16; i += 4)
  {
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    z = rj_xtime(e);
    x = e ^ rj_xtime(rj_xtime(z^a^c));
    y = e ^ rj_xtime(rj_xtime(z^b^d));
    buf[i] ^= x ^ rj_xtime(a^b);
    buf[i+1] ^= y ^ rj_xtime(b^c);
    buf[i+2] ^= x ^ rj_xtime(c^d);
    buf[i+3] ^= y ^ rj_xtime(d^a);
  }
} /* aes_mixColumns_inv */

/* -------------------------------------------------------------------------- */
__device__ __host__ void aes_expandEncKey(uint8_t *k, uint8_t *rc, const uint8_t *sb) 
{
  register uint8_t i;

  k[0] ^= sb[k[29]] ^ (*rc);
  k[1] ^= sb[k[30]];
  k[2] ^= sb[k[31]];
  k[3] ^= sb[k[28]];
  *rc = F( *rc);

  for(i = 4; i < 16; i += 4)
  {
    k[i] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }
  
  k[16] ^= sb[k[12]];
  k[17] ^= sb[k[13]];
  k[18] ^= sb[k[14]];
  k[19] ^= sb[k[15]];

  for(i = 20; i < 32; i += 4)
  {
    k[i] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

} /* aes_expandEncKey */

/* -------------------------------------------------------------------------- */
__device__ void aes_expandDecKey(uint8_t *k, uint8_t *rc) 
{
  uint8_t i;

  for(i = 28; i > 16; i -= 4)
  {
    k[i+0] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

  k[16] ^= sbox_d[k[12]];
  k[17] ^= sbox_d[k[13]];
  k[18] ^= sbox_d[k[14]];
  k[19] ^= sbox_d[k[15]];

  for(i = 12; i > 0; i -= 4)
  {
    k[i+0] ^= k[i-4];
    k[i+1] ^= k[i-3];
    k[i+2] ^= k[i-2];
    k[i+3] ^= k[i-1];
  }

  *rc = FD(*rc);
  k[0] ^= sbox_d[k[29]] ^ (*rc);
  k[1] ^= sbox_d[k[30]];
  k[2] ^= sbox_d[k[31]];
  k[3] ^= sbox_d[k[28]];
} /* aes_expandDecKey */

/* -------------------------------------------------------------------------- */
void aes256_init(uint8_t *k)
{
  uint8_t rcon = 1;
  register uint8_t i;

  for (i = 0; i < sizeof(ctx_key); i++)
  {
    ctx_enckey[i] = ctx_deckey[i] = k[i];
  }
  for (i = 8;--i;)
  {
    aes_expandEncKey(ctx_deckey, &rcon, sbox);
  }
} /* aes256_init */

/* -------------------------------------------------------------------------- */
__global__ void aes256_encrypt_ecb(uint8_t *buf_d, unsigned long numbytes, uint8_t *ctx_enckey_d, uint8_t *ctx_key_d)
{
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE]; // thread buffer
  //printf("Thread %d\n", threadIdx.x);
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) { 
    //printf("Thread %d exiting\n", threadIdx.x);
    return; }
  //printf("Thread %d working at offset %lu\n", threadIdx.x, offset);
  /* copy data from global memory into thread buffer */
  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_enckey_d, ctx_key_d);
  for(i = 1, rcon = 1; i < 14; ++i)
  {
    aes_subBytes(buf_t);
    aes_shiftRows(buf_t);
    aes_mixColumns(buf_t);
    if( i & 1 )
    {
      aes_addRoundKey( buf_t, &ctx_key_d[16]);
    }
    else
    {
      aes_expandEncKey(ctx_key_d, &rcon, sbox_d), aes_addRoundKey(buf_t, ctx_key_d);
    }
  }
  aes_subBytes(buf_t);
  aes_shiftRows(buf_t);
  aes_expandEncKey(ctx_key_d, &rcon, sbox_d);
  aes_addRoundKey(buf_t, ctx_key_d);
  /* copy thread buffer back into global memory */
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} /* aes256_encrypt */

/* -------------------------------------------------------------------------- */
__global__ void aes256_decrypt_ecb(uint8_t *buf_d, unsigned long numbytes, uint8_t *ctx_deckey_d, uint8_t *ctx_key_d)
{
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE]; // thread buffer
  //printf("Thread %d\n", threadIdx.x);
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) {
    //printf("Thread %d exiting\n", threadIdx.x);
    return; }
  //printf("Thread %d working at offset %lu\n", threadIdx.x, offset);
  /* copy data from global memory into thread buffer */
  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_deckey_d, ctx_key_d);
  aes_shiftRows_inv(buf_t);
  aes_subBytes_inv(buf_t);
  for (i = 14, rcon = 0x80; --i;)
  {
    if( ( i & 1 ) )
    {
      aes_expandDecKey(ctx_key_d, &rcon);
      aes_addRoundKey(buf_t, &ctx_key_d[16]);
    }
    else
    {
      aes_addRoundKey(buf_t, ctx_key_d);
    }
    aes_mixColumns_inv(buf_t);
    aes_shiftRows_inv(buf_t);
    aes_subBytes_inv(buf_t);
    }
  aes_addRoundKey( buf_t, ctx_key_d);
  /* copy thread back into global memory */
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} /* aes256_decrypt */
/******************************************************************/
void encrypt(uint8_t key[32], uint8_t *buf, unsigned long numbytes)
{
  uint8_t *buf_d;
  uint8_t *ctx_key_d, *ctx_enckey_d;

  hipMemcpyToSymbol(HIP_SYMBOL(sbox_d), sbox, sizeof(uint8_t)*256);

  printf("\nBeginning encryption\n");
  aes256_init(key);
  hipMalloc((void**)&buf_d, numbytes);
  hipMalloc((void**)&ctx_enckey_d, sizeof(ctx_enckey));
  hipMalloc((void**)&ctx_key_d, sizeof(ctx_key));
  hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(ctx_enckey_d, ctx_enckey, sizeof(ctx_enckey), hipMemcpyHostToDevice);
  hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice);
  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  aes256_encrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes, ctx_enckey_d, ctx_key_d);

  hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);
  hipMemcpy(ctx_enckey, ctx_enckey_d, sizeof(ctx_enckey), hipMemcpyDeviceToHost);
  hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost);

  hipFree(buf_d);
  hipFree(ctx_key_d);
  hipFree(ctx_enckey_d);
}
/******************************************************************/
void decrypt(uint8_t key[32], uint8_t *buf, unsigned long numbytes)
{
  uint8_t *buf_d;
  uint8_t *ctx_key_d, *ctx_deckey_d;

  hipMemcpyToSymbol(HIP_SYMBOL(sboxinv_d), sboxinv, sizeof(uint8_t)*256);

  printf("\nBeginning decryption\n");

  hipMalloc((void**)&buf_d, numbytes);
  hipMalloc((void**)&ctx_deckey_d, sizeof(ctx_deckey));
  hipMalloc((void**)&ctx_key_d, sizeof(ctx_key));

  hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(ctx_deckey_d, ctx_deckey, sizeof(ctx_deckey), hipMemcpyHostToDevice);
  hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice);
  
  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  printf("Creating %d threads over %d blocks\n", dimBlock.x*dimGrid.x, dimBlock.x);
  aes256_decrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes, ctx_deckey_d, ctx_key_d);

  hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);
  hipMemcpy(ctx_deckey, ctx_deckey_d, sizeof(ctx_deckey), hipMemcpyDeviceToHost);
  hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost);

  hipFree(buf_d);
  hipFree(ctx_key_d);
  hipFree(ctx_deckey_d);
}

__global__ void GPU_init() { }


/******************************************************************/



extern "C" char* gpu_encrypt (char *akey, char *plain)
{
  printf("2.  lenth is %d\n",strlen((char*)akey));
//  FILE *file,*keyfile;
  uint8_t *buf; // file buffer
  unsigned long numbytes;
  int keylenth,plainlenth;
//  char *fname,*keyfilename;
  int i;
  int padding;
  //key: 00 01 02 03 04 05 06 07 08 09 0a 0b 0c 0d 0e 0f 10 11 12 13 14 15 16 17 18 19 1a 1b 1c 1d 1e 1f
    /* create a key vector */
  uint8_t key[32];

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(error_id));
    printf("Exiting...\n");
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("There are no available device(s) that support CUDA\n");
    exit(EXIT_FAILURE);
  }
 // generate key
/*  if(argc < 2)
  {
    printf("Please input the key filename");
    exit(1);
  }  
else
    keyfilename = argv[1];
*/
//  keyfile = fopen(keyfilename,"r");
//  if (keyfile == NULL) {printf("keyfile %s doesn't exist\n", keyfilename); exit(1); }
  keylenth = strlen(akey);
  for (i = 0; i < 32;i++)
  {
    char ch;
    ch = akey[i];
    key[i] = ch;
  }
  for(int i = 0; i < 32; i++)
    printf("%c ", key[i]);
  printf("\n************************************\n");
  for(int i = 0; i < strlen(plain); i++)
    printf("%c ", plain[i]);
  plainlenth = strlen(plain);
  printf("\nplain lenth is %d\n",strlen(plain));
  numbytes = plainlenth + 4;
  
  // calculate the padding
  padding =AES_BLOCK_SIZE  - numbytes % AES_BLOCK_SIZE;
  numbytes += padding;

  // copy file into memory
  buf = (uint8_t*)calloc(numbytes, sizeof(uint8_t));
  for(int i = 0; i < plainlenth; i++)
    buf[i] = plain[i];
  int buflenth;
  buflenth = strlen((char*)buf);
  unsigned char *intp;
  int *padp;
  padp = &padding;
  intp = (unsigned char*) padp;
  for(int k = 0; k < 4; k++)
  { 
    buf[buflenth++] = (unsigned char)*intp;
    intp += 1;
  }  
  // this is to force nvcc to put the gpu initialization here
  GPU_init<<<1, 1>>>();
  
/******************************************************************/
  encrypt(key, buf, numbytes);
/*  file = fopen("cipher.txt", "w");
  fwrite(buf, 1, numbytes, file);
  fwrite(&padding, 4, 1, file);
  fclose(file);
  free(buf);
*/
/*  char *cipher;
  printf("buflenth is %d \n",buflenth);
  for(int k = 0; k < buflenth; k++)
    cipher[k] = buf[k];
*/
/*  int buflenth;
  buflenth = strlen((char*)buf);
  unsigned char *intp;
  int *padp;
  padp = &padding;
  intp = (unsigned char*) padp;
  for(int k = 0; k < 4; k++)
  { 
    printf("buflenth = %d   ",buflenth);
    printf("cipher lenth is %d \n",strlen((char*)buf)); 
    buf[buflenth++] = *intp;
    intp += 1;
  }  
  printf("finished!\n");
  buf[17] = *intp;
  printf("cipher lenth is %d \n",strlen((char*)buf));
*/
  return (char*)buf;
} /* main */

